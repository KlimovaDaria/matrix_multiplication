#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <hip/device_functions.h>
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <ctime>




#define BLOCK_SIZE 32 

__global__ void kernel(float* a, float* b, int n, float* c) //ГПУ
{
	int bx = blockIdx.x;
	int by = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	float sum = 0.0f;
	int ia = n * BLOCK_SIZE * by + n * ty; // a [i][0]
	int ib = BLOCK_SIZE * bx + tx;


	for (int k = 0; k < n; k++)
		sum += a[ia + k] * b[ib + k * n]; 

	int ic = n * BLOCK_SIZE * by + BLOCK_SIZE * bx;

	c[ic + n * ty + tx] = sum;
}



int main()
{
	int N = 1000; // размер квадратной матрицы
	int m, n, k;

	float timerValueGPU = 0.0f;
	float timerValueCPU = 0.0f;
	hipEvent_t start;
	hipEvent_t stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipError_t err;

	int numBytes = N * N * sizeof(float);
	float* adev, * bdev, * cdev, * a, * b, * c, * cc;

	a = (float*)malloc(numBytes);
	b = (float*)malloc(numBytes);
	c = (float*)malloc(numBytes);
	cc = (float*)malloc(numBytes);



	for (n = 0; n < N; n++)
		for (m = 0; m < N; m++) {
			a[m + n * N] = 2.0f * m + n;
			b[m + n * N] = 2.0f * n;
		}

	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	dim3 blocks(N / threads.x, N / threads.y);

	hipMalloc((void**)&adev, numBytes);
	hipMalloc((void**)&bdev, numBytes);
	hipMalloc((void**)&cdev, numBytes);


	hipEventRecord(start, 0);

	hipMemcpy(adev, a, numBytes, hipMemcpyHostToDevice);
	hipMemcpy(bdev, b, numBytes, hipMemcpyHostToDevice);

	kernel << < blocks, threads >> > (adev, bdev, N, cdev);

	err = hipPeekAtLastError();
	if (err != hipSuccess)
		printf(hipGetErrorString(err));

	hipMemcpy(c, cdev, numBytes, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&timerValueGPU, start, stop);
	printf("\n GPU calculation time %f msec\n", timerValueGPU);





	clock_t startc;
	clock_t stopc;

	startc = clock();

	for (n = 0; n < N; n++)
		for (m = 0; m < N; m++) {
			cc[m + n * N] = 0.f;
			for (k = 0; k < N; k++)
				cc[m + n * N] += a[k + n * N] * b[k * N + m];
		}

	stopc = clock();

	printf("\n CPU calculation time %f sec\n", ((double)(stopc - startc)) / ((double)CLOCKS_PER_SEC));

	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipFree(&adev);
	hipFree(&bdev);
	hipFree(&cdev);

	delete a;
	delete b;
	delete c;
	delete cc;

	return 0;
}